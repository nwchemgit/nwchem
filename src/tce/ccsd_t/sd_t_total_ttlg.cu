#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

__device__ double* t3_s_d;
__device__ double* t3_d;

#include "header.h"
#include "ourinclude.h"

#define T1 16
#define T2 16
#define Tcomm 16

hipblasHandle_t handle;
double* output_d;
size_t current_i_size;
extern "C" void ttgt_init()
{
  hipblasCreate(&handle);
  output_d = NULL;
  current_i_size = 0;
}
extern    "C" void set_dev_mem_d(int h1d, int h2d, int h3d, int p4d, int p5d,int p6d)
{
    int size_t3;
    size_t3 = h1d*h2d*h3d*p4d*p5d*p6d;
    t3_d = (double *) getGpuMem(size_t3*sizeof(double));
    hipMemset(t3_d,0,size_t3*sizeof(double));
}
extern          "C" void
dev_mem_d_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d)
{
    set_dev_mem_d((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d);
}
extern "C" void
dev_release()
{

        freeGpuMem(t3_d);
        freeGpuMem(t3_s_d);
}
extern "C" void
dev_release_()
{
    dev_release();
}
extern "C" void sd_t_d1_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub, int id) {
double* output_d;
	static int count = 0;
	if(count == 0)
	{
		ttgt_init();
		count++;
	}

	size_t size_triplesx,size_block_triplesx,size_el_block_triplesx,size_t2sub,size_v2sub;
        size_t i;
        double *t2sub_d,*v2sub_d;
        size_triplesx= p4d * p5d * h1d * h3d * h2d * p6d *sizeof(double);
        size_t2sub=h7d*p4d*p5d*h1d*sizeof(double);
        size_v2sub=h3d*h2d*p6d*h7d*sizeof(double);
        int i1[4], i2[4], o[6];
        i1[0] = h7d;
        i1[1] = p4d;
        i1[2] = p5d;
        i1[3] = h1d;
        i2[0] = h3d;
        i2[1] = h2d;
        i2[2] = p6d;
        i2[3] = h7d;
        o[0] = p4d;
        o[1] = p5d;
        o[2] = h1d;
        o[3] = h3d;
        o[4] = h2d;
        o[5] = p6d;
        hipblasOperation_t transa, transb;
        transa = HIPBLAS_OP_T;
        transb = HIPBLAS_OP_T;
        size_t m,n,k;
        m = p4d*p5d*h1d;
        k = h7d;
        n = h3d*h2d*p6d;
        double alpha, beta;
        alpha = 1;
        beta = 0;
        t2sub_d=(double*)getGpuMem(size_t2sub);
        v2sub_d=(double*)getGpuMem(size_v2sub);
	//if(size_triplesx > current_i_size)
	{
        	output_d=(double*)getGpuMem(size_triplesx);
		current_i_size = size_triplesx;
	}
	if(output_d == NULL) 
	{
		exit(0);
	}
        int perm[6];
	//double beta;
	switch(id)
	{
		case 1:
        		perm[0] = 3;
		        perm[1] = 4;
		        perm[2] = 2;
		        perm[3] = 5;
		        perm[4] = 1;
		        perm[5] = 0;
			beta = -1.0;
		break;
		case 2:
        		perm[0] = 3;
		        perm[1] = 2;
		        perm[2] = 4;
		        perm[3] = 5;
		        perm[4] = 1;
		        perm[5] = 0;
			beta = 1.0;
		break;
		case 3:
        		perm[0] = 2;
		        perm[1] = 3;
		        perm[2] = 4;
		        perm[3] = 5;
		        perm[4] = 1;
		        perm[5] = 0;
			beta = -1.0;
		break;
		case 4:
        		perm[0] = 3;
		        perm[1] = 4;
		        perm[2] = 2;
		        perm[3] = 1;
		        perm[4] = 0;
		        perm[5] = 5;
			beta = -1.0;
		break;
		case 5:
        		perm[0] = 3;
		        perm[1] = 2;
		        perm[2] = 4;
		        perm[3] = 1;
		        perm[4] = 0;
		        perm[5] = 5;
			beta = 1.0;
		break;
		case 6:
        		perm[0] = 2;
		        perm[1] = 3;
		        perm[2] = 4;
		        perm[3] = 1;
		        perm[4] = 0;
		        perm[5] = 5;
			beta = -1.0;
		break;
		case 7:
        		perm[0] = 3;
		        perm[1] = 4;
		        perm[2] = 2;
		        perm[3] = 1;
		        perm[4] = 5;
		        perm[5] = 0;
			beta = 1.0;
		break;
		case 8:
        		perm[0] = 3;
		        perm[1] = 2;
		        perm[2] = 4;
		        perm[3] = 1;
		        perm[4] = 5;
		        perm[5] = 0;
			beta = -1.0;
		break;
		case 9:
        		perm[0] = 2;
		        perm[1] = 3;
		        perm[2] = 4;
		        perm[3] = 1;
		        perm[4] = 5;
		        perm[5] = 0;
			beta = 1.0;
		break;
	}

        hipblasDgemm(handle, transa, transb, m, n, k, &alpha, t2sub_d, h7d, v2sub_d, n, &beta, output_d, m);
       ttlg_transpose(6, o, perm, output_d, t3_d, 1, beta);
        hipDeviceSynchronize();
        freeGpuMem(t2sub_d);
        freeGpuMem(v2sub_d);
  	freeGpuMem(output_d);
}

/*----------------------------------------------------------------------*
 *triplesx[h3,h1,p6,p5,p4] -= t2sub[h7,p4,p5,h1] * v2sub[h3,p6,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_1_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 1);
}

extern "C" void sd_t_d1_1_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_1_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h3,h1,h2,p5,p4] += t2sub[h7,p4,p5,h1] * v2sub[h3,h2,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_2_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 2);
}
extern "C" void sd_t_d1_2_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_2_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h1,h3,p5,p4] -= t2sub[h7,p4,p5,h1] * v2sub[h3,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_3_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 3);
}
extern "C" void sd_t_d1_3_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_3_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h3,h1,p5,p4,p6] -= t2sub[h7,p4,p5,h1] * v2sub[h3,p6,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_4_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 4);
}
extern "C" void sd_t_d1_4_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_4_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h3,h1,h2,p5,p4,p6] += t2sub[h7,p4,p5,h1] * v2sub[h3,h2,p6,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_5_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 5);
}
extern "C" void sd_t_d1_5_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_5_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h1,h3,p5,p4,p6] -= t2sub[h7,p4,p5,h1] * v2sub[h3,p6,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_6_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 6);
}
extern "C" void sd_t_d1_6_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_6_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h3,h1,p5,p6,p4] += t2sub[h7,p4,p5,h1] * v2sub[h3,p6,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_7_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 7);
}
extern "C" void sd_t_d1_7_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_7_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h3,h1,h2,p5,p6,p4] -= t2sub[h7,p4,p5,h1] * v2sub[h3,h2,p6,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_8_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 8);
}
extern "C" void sd_t_d1_8_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_8_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}
/*----------------------------------------------------------------------*
 *triplesx[h1,h3,p5,p6,p4] += t2sub[h7,p4,p5,h1] * v2sub[h3,p6,h7]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d1_9_cuda(int h1d, int h2d, int h3d, int h7d, int p4d, int p5d, int p6d, double *triplesx, double *t2sub, double *v2sub) {
sd_t_d1_cuda( h1d,  h2d,  h3d,  h7d,  p4d,  p5d,  p6d, triplesx, t2sub, v2sub, 9);
}
extern "C" void sd_t_d1_9_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* h7d, Integer* p4d, Integer* p5d, Integer* p6d, double *triplesx, double *t2sub, double *v2sub) {
  sd_t_d1_9_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*h7d,(int)*p4d,(int)*p5d,(int)*p6d,triplesx,t2sub,v2sub);
}


extern "C" void sd_t_d2_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *triplesx, double *t2sub, double *v2sub, int id) {
 size_t size_triplesx,size_block_triplesx,size_el_block_triplesx,size_t2sub,size_v2sub;
double* output_d;
        size_t i;
        double *t2sub_d,*v2sub_d;
        size_triplesx= p4d * p5d * h1d * h3d * h2d * p6d *sizeof(double);
        size_t2sub=p7d*p4d*p5d*h1d*sizeof(double);
        size_v2sub=h3d*h2d*p6d*p7d*sizeof(double);
        int i1[4], i2[4], o[6];
        i1[0] = p7d;
        i1[1] = p4d;
        i1[2] = h1d;
        i1[3] = h2d;
        i2[0] = p7d;
        i2[1] = h3d;
        i2[2] = p6d;
        i2[3] = p5d;
        o[0] = p4d;
        o[1] = h1d;
        o[2] = h2d;
        o[3] = h3d;
        o[4] = p6d;
        o[5] = p5d;
        hipblasOperation_t transa, transb;
        transa = HIPBLAS_OP_T;
        transb = HIPBLAS_OP_N;
        size_t m,n,k;
        m = p4d*h1d*h2d;
        k = p7d;
        n = h3d*p6d*p5d;
        double alpha, beta;
        alpha = 1;
        beta = 0;
        t2sub_d=(double*)getGpuMem(size_t2sub);
        v2sub_d=(double*)getGpuMem(size_v2sub);
	//if(size_triplesx > current_i_size)
	{
        	output_d=(double*)getGpuMem(size_triplesx);
		current_i_size = size_triplesx;
	}
	if(output_d == NULL) 
	{
		exit(0);
	}
        int perm[6];
	//double beta;
	switch(id)
	{
		case 1:
        		perm[0] = 3;
		        perm[1] = 2;
		        perm[2] = 1;
		        perm[3] = 4;
		        perm[4] = 5;
		        perm[5] = 0;
			beta = -1.0;
		break;
		case 2:
        		perm[0] = 2;
		        perm[1] = 1;
		        perm[2] = 3;
		        perm[3] = 4;
		        perm[4] = 5;
		        perm[5] = 0;
			beta = -1.0;
		break;
		case 3:
        		perm[0] = 2;
		        perm[1] = 3;
		        perm[2] = 1;
		        perm[3] = 4;
		        perm[4] = 5;
		        perm[5] = 0;
			beta = 1.0;
		break;
		case 4:
        		perm[0] = 3;
		        perm[1] = 2;
		        perm[2] = 1;
		        perm[3] = 4;
		        perm[4] = 0;
		        perm[5] = 5;
			beta = 1.0;
		break;
		case 5:
        		perm[0] = 2;
		        perm[1] = 1;
		        perm[2] = 3;
		        perm[3] = 4;
		        perm[4] = 0;
		        perm[5] = 5;
			beta = 1.0;
		break;
		case 6:
        		perm[0] = 2;
		        perm[1] = 3;
		        perm[2] = 1;
		        perm[3] = 4;
		        perm[4] = 0;
		        perm[5] = 5;
			beta = -1.0;
		break;
		case 7:
        		perm[0] = 3;
		        perm[1] = 2;
		        perm[2] = 1;
		        perm[3] = 0;
		        perm[4] = 4;
		        perm[5] = 5;
			beta = -1.0;
		break;
		case 8:
        		perm[0] = 2;
		        perm[1] = 1;
		        perm[2] = 3;
		        perm[3] = 0;
		        perm[4] = 4;
		        perm[5] = 5;
			beta = -1.0;
		break;
		case 9:
        		perm[0] = 2;
		        perm[1] = 3;
		        perm[2] = 1;
		        perm[3] = 0;
		        perm[4] = 4;
		        perm[5] = 5;
			beta = 1.0;
		break;
	}

        hipblasDgemm(handle, transa, transb, m, n, k, &alpha, t2sub_d, p7d, v2sub_d, n, &beta, output_d, m);
     ttlg_transpose(6, o, perm, output_d, t3_d, 1, beta);
        hipDeviceSynchronize();
        freeGpuMem(t2sub_d);
        freeGpuMem(v2sub_d);
        	freeGpuMem(output_d);
}

/*----------------------------------------------------------------------*
 *t3[h3,h2,h1,p6,p4] -= t2[p7,p4,h1,h2] * v2[p7,h3,p6]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_1_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 1);
}
extern "C" void sd_t_d2_1_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_1_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h2,h1,h3,p4] -= t2[p7,p4,h1,h2] * v2[p7,h3]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_2_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 2);
}
extern "C" void sd_t_d2_2_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_2_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h2,h3,h1,p6,p4] += t2[p7,p4,h1,h2] * v2[p7,h3,p6]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_3_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 3);
}
extern "C" void sd_t_d2_3_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_3_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h3,h2,h1,p6,p4,p5] += t2[p7,p4,h1,h2] * v2[p7,h3,p6,p5]
 *----------------------------------------------------------------------*/

extern "C" void sd_t_d2_4_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 4);
}
extern "C" void sd_t_d2_4_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_4_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h2,h1,h3,p4,p5] += t2[p7,p4,h1,h2] * v2[p7,h3,p5]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_5_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 5);
}
extern "C" void sd_t_d2_5_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_5_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h2,h3,h1,p6,p4,p5] -= t2[p7,p4,h1,h2] * v2[p7,h3,p6,p5]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_6_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 6);
}
extern "C" void sd_t_d2_6_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_6_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h3,h2,h1,p4,p6,p5] -= t2[p7,p4,h1,h2] * v2[p7,h3,p6,p5]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_7_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 7);
}
extern "C" void sd_t_d2_7_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_7_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h2,h1,h3,p4,p6,p5] -= t2[p7,p4,h1,h2] * v2[p7,h3,p6,p5]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_8_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 8);
}
extern "C" void sd_t_d2_8_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_8_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}
/*----------------------------------------------------------------------*
 *t3[h2,h3,h1,p4,p6,p5] += t2[p7,p4,h1,h2] * v2[p7,h3,p6,p5]
 *----------------------------------------------------------------------*/
extern "C" void sd_t_d2_9_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p7d, double *t3, double *t2, double *v2) {
	sd_t_d2_cuda(h1d, h2d, h3d, p4d,  p5d, p6d,  p7d, t3, t2, v2, 9);
}

extern "C" void sd_t_d2_9_cuda_(Integer *h1d, Integer* h2d, Integer* h3d, Integer* p4d, Integer* p5d, Integer* p6d, Integer* p7d, double *t3, double *t2, double *v2) {
  sd_t_d2_9_cuda((int)*h1d,(int)*h2d,(int)*h3d,(int)*p4d,(int)*p5d,(int)*p6d,(int)*p7d,t3,t2,v2);
}


#define MAX_h3 64
/* IMPORTANT!!!!
t3_d must be passed as parameter to kernel function. A __global__ function can't access the global variable directly*/

__global__ void compute_energy_kernel(int h1d,int h2d,int h3d,int p4d,int p5d,int p6d,double* eval1,double* eval2,double* eval3,double* eval4,double* eval5,double* eval6, double* energy, double factor, int total_size, double* t3d, double* t3_sd)
{
  int h1,h2,p6,p4,p5, h3,i=0;
  double e1,e2,e4,e5,e6;
//  __shared__ double t2_shm[MAX_h3];
  __shared__ double energy_s[T1];
  __shared__ double energy2_s[T1];
  double inner_fac;
  int limit;
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  if(threadIdx.x==0)
  {
        energy[blockIdx.x]=0;
        energy[blockIdx.x+gridDim.x]=0;
        energy_s[threadIdx.x] = 0.0;
        energy2_s[threadIdx.x] = 0.0;
  }

  for(int j =0; j<T2*T1;j++) {
    thread_x = T2*T1*blockIdx.x + j;  
    rest_x = thread_x;
    __syncthreads();
    h2=rest_x%h2d;
    rest_x=rest_x/h2d;
    h1=rest_x%h1d;
    rest_x=rest_x/h1d;
    p6=rest_x%p6d;
    rest_x=rest_x/p6d;
    p5=rest_x%p5d;
    rest_x=rest_x/p5d;
    p4=rest_x%p4d;
    e1 = eval1[h1];
    e2 = eval2[h2];
    e4 = eval4[p4];
    e5 = eval5[p5];
    e6 = eval6[p6];
/*
  for(p4=0;p4<p4d;p4++) 
    for(p5 = 0;p5<p5d;p5++)
        for(p6=0;p6<p6d;p6++) 
            for(h1= 0;h1<h1d;h1++) 
                for(h2=0;h2<h2d;h2++) 
                    for(h3=0;h3<h3d;h3++) {
                        inner_fac = -eval4[p4]-eval5[p5]-eval6[p6]+eval1[h1]
                            +eval2[h2]+eval3[h3];
                        energy_s[0]+=factor*t3d[i]*t3d[i]/inner_fac;
                        energy2_s[0]+=factor*t3d[i]*(t3_sd[i]+t3d[i])/inner_fac;
                        i++;
                    }
*/
    if(thread_x<total_size)
    for(int i=0;i<h3d;i++)
    {
        inner_fac = -e4-e5-e6+e1+e2+eval3[i]; //t2_shm[i];
//ckbn avoid e1 in case we need just (T)
        energy_s[threadIdx.x] += factor* t3d[thread_x*h3d+i]*t3d[thread_x*h3d+i]/inner_fac;
        energy2_s[threadIdx.x] += factor* t3d[thread_x*h3d+i]*(t3_sd[thread_x*h3d+i]+t3d[thread_x*h3d+i])/inner_fac;
    }
    __syncthreads();
  }
  if(threadIdx.x==0)
  {
/*	  limit = blockDim.x;
      if (blockIdx.x == (gridDim.x-1)) limit = total_size%blockDim.x;
      for(int i=0;i<limit;i++)
      {
        energy[blockIdx.x]+=energy_s[i];
        energy[blockIdx.x+gridDim.x]+=energy2_s[i];
      }
*/
    energy[blockIdx.x] = energy_s[0];
    energy[blockIdx.x+gridDim.x] = energy2_s[0];
   }
  __syncthreads();

}

extern   "C" void compute_energy(double factor, double* energy, double* eval1, double* eval2,double* eval3,double* eval4,double* eval5,double* eval6,int h1d, int h2d, int h3d, int p4d, int p5d,int p6d, double* host1, double* host2)
//ckbn en_comment, double* total_d, double* total_s)
{
    double* energy_d, *energy_h;
    double* eval_d1,*eval_d2,*eval_d3,*eval_d4,*eval_d5,*eval_d6;
    int size_energy = 2*sizeof(double);
    int total_block = DIV_UB((h1d*h2d*p4d*p5d*p6d), (T2*T1));

//    int total_block = 1;
    int total_elements = h1d*h2d*p4d*p5d*p6d;

    energy_d = (double*)getGpuMem(size_energy*total_block*2);
    int i=0,in; 
    double* t3 = (double*)malloc(sizeof(double)*h3d*total_elements);
    double* ts3 = (double*)malloc(sizeof(double)*h3d*total_elements);

    energy_h = (double*)getHostMem(size_energy*2*total_block);
    eval_d1 = (double*)getGpuMem(h1d*sizeof(double));
    eval_d2 = (double*)getGpuMem(h2d*sizeof(double));
    eval_d3 = (double*)getGpuMem(h3d*sizeof(double));
    eval_d4 = (double*)getGpuMem(p4d*sizeof(double));
    eval_d5 = (double*)getGpuMem(p5d*sizeof(double));
    eval_d6 = (double*)getGpuMem(p6d*sizeof(double));

    CUDA_SAFE(hipMemcpy(eval_d1, eval1, h1d*sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE(hipMemcpy(eval_d2, eval2, h2d*sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE(hipMemcpy(eval_d3, eval3, h3d*sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE(hipMemcpy(eval_d4, eval4, p4d*sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE(hipMemcpy(eval_d5, eval5, p5d*sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE(hipMemcpy(eval_d6, eval6, p6d*sizeof(double), hipMemcpyHostToDevice));
/* for test only */
//printf("host 2 is %f %f\n", host2[0], host2[1]);
//    CUDA_SAFE(hipMemcpy(t3_s_d, host2, total_elements*h3d*sizeof(double), hipMemcpyHostToDevice));

    dim3 dimBlock(1); //T2*T1);
    dim3 dimGrid(total_block);
    compute_energy_kernel<<<dimGrid,dimBlock,0>>>(h1d,h2d,h3d,p4d,p5d,p6d, eval_d1,eval_d2,eval_d3,eval_d4,eval_d5,eval_d6,energy_d, factor, h1d*h2d*p4d*p5d*p6d, t3_d, t3_s_d);
	hipDeviceSynchronize();
    //CHECK_ERR("Kernel execution failed");
    CUDA_SAFE(hipMemcpy(((char *) energy_h) , ((char *) energy_d) , 
    size_energy*total_block*2, hipMemcpyDeviceToHost));

    for(int i=1;i<dimGrid.x;i++)
      {
        energy_h[0]+=energy_h[i];
        energy_h[dimGrid.x]+=energy_h[i+dimGrid.x];
      }

     
//    printf("CUDA energy_h is %f %f %d %d %d %d %d %d\n", energy_h[0], energy_h[dimGrid.x]); //, total_size, h1d, h2d, p4d, p5d,p6d);
/*
    CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_d) , sizeof(double)*h3d*total_elements, hipMemcpyDeviceToHost));
    CUDA_SAFE(hipMemcpy(((char *) ts3) , ((char *) t3_s_d) , sizeof(double)*h3d*total_elements, hipMemcpyDeviceToHost));
    total_s[0]=0.0, total_d[0]=0.0;
    for(int i=0;i<h3d*total_elements;i++) {
        total_s[0] += ts3[i];
        total_d[0] += t3[i];
    }
*/
//    printf("Total doubles and singles %f, %f\n", total_d, total_s);
    energy[0] = energy_h[0];
    energy[1] = energy_h[dimGrid.x];
    freeGpuMem(energy_d);
    freeGpuMem(eval_d1);
    freeGpuMem(eval_d2);
    freeGpuMem(eval_d3);
    freeGpuMem(eval_d4);
    freeGpuMem(eval_d5);
    freeGpuMem(eval_d6);
    freeHostMem(energy_h);
}
extern          "C" void
compute_en_(double * factor, double * energy, double * eval1,double* eval2,double* eval3,double* eval4,double* eval5,double* eval6, Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d, double* host1, double* host2)
//ckbn en_comment,double* total_d, double* total_s)
{
    compute_energy((double) *factor, energy, eval1,eval2, eval3, eval4, eval5, eval6,(int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d, host1, host2);
//ckbn en_comment    ,total_d, total_s);
}

//__device__ double* t3_d; 
extern    "C" void set_dev_mem_s(int h1d, int h2d, int h3d, int p4d, int p5d,int p6d)
{
    int size_t3;
    size_t3 = h1d*h2d*h3d*p4d*p5d*p6d;
    t3_s_d = (double *) getGpuMem(size_t3*sizeof(double));
    hipMemset(t3_s_d,0,size_t3*sizeof(double));
}



extern          "C" void
dev_mem_s_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d)
{
    set_dev_mem_s((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d);
}

/*----------------------------------------------------------------------*
 *t3[h3,h2,h1,p6,p5,p4] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
#define T1 16
#define T2 16
#define Tcomm 16
__global__ void sd_t_s1_1_kernel(int h1d,int h2d,int h3d,int p4d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p4ld_t3, double *t2_d, double *v2_d,int p4, int total_x, double* t3d) {
  int h1,h2,h3,p6;
  __shared__ double t2_shm[T1*2*Tcomm];
  
  for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
  if(i<h1d*p4d)
  t2_shm[i] = t2_d[i];
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  rest_x = thread_x;
    __syncthreads();
/* the following computation may need to happen inside the loop */
  for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
  {
    rest_x += i;
  	h3=rest_x%h3d;
  	rest_x=rest_x/h3d;
  	h2=rest_x%h2d;
  	rest_x=rest_x/h2d;
  	p6=rest_x%p6d;

    if((thread_x+i)<total_x)
  	for(h1=0;h1<h1d;h1++)
  	for(p4=0;p4<p4d;p4++)
  	{
     	t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p4*p4ld_t3]+=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2];
  	}
  }
    __syncthreads();
}

extern          "C" void 
sd_t_s1_1_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double         *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
//CUDA_SAFE(hipMalloc((void**) &t3_d, size_t3));
//CUDA_SAFE(hipMalloc((void**) &t2_d, size_t2));
//CUDA_SAFE(hipMalloc((void**) &v2_d, size_v2));
//	t3_d = (double *) getGpuMem(size_t3);
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
//	p5ld_v2 = p6d * h3d * p7d;
	h3ld_t3 = 1;
	h2ld_t3 = h3d;
	h1ld_t3 = h2d * h3d;
	p6ld_t3 = h1d * h2d * h3d;
//	p5ld_t3 = p6d * h1d * h2d * h3d;
	p4ld_t3 = p5d * p6d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
  for(i=0;i<nstreams;++i){
    sd_t_s1_1_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d*p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t2_d,v2_d,i,total_x, t3_s_d);
		CHECK_ERR("Kernel execution failed");
	}
/*
    st = timer();
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}

	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}
*/
	hipDeviceSynchronize();

//	CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));
	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
//	freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
   //  hipFree(t2_d);
   //  hipFree(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
#undef T1
#undef T2
#undef Tcomm
extern          "C" void 
sd_t_s1_1_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d, double *t3, double *t2, double *v2)
{
	sd_t_s1_1_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d,  t3, t2, v2);
}
/*----------------------------------------------------------------------*
 *t3[h3,h1,h2,p6,p5,p4] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
#define T1 16
#define T2 16
#define Tcomm 16
__global__ void sd_t_s1_2_kernel(int h1d,int h2d,int h3d,int p4d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p4ld_t3,double *t2_d, double *v2_d,int p4, int total_x, double* t3d) {
  int h1,h2,h3,p6;
  __shared__ double t2_shm[T1*2*Tcomm];
  
  for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
  if(i<h1d*p4d)
  t2_shm[i] = t2_d[i];
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  rest_x = thread_x;
    __syncthreads();
/* the following computation may need to happen inside the loop */
  for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
  {
    rest_x += i;
  	h3=rest_x%h3d;
  	rest_x=rest_x/h3d;
  	h2=rest_x%h2d;
  	rest_x=rest_x/h2d;
  	p6=rest_x%p6d;

    if((thread_x+i)<total_x)
  	for(h1=0;h1<h1d;h1++)
  	for(p4=0;p4<p4d;p4++)
  	{
     	t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p4*p4ld_t3]-=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2];
  	}
  }
    __syncthreads();
}

extern          "C" void 
sd_t_s1_2_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double         *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
/*    if(first==1)
    {
		t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
	}*/
//CUDA_SAFE(hipMalloc((void**) &t2_d, size_t2));
//CUDA_SAFE(hipMalloc((void**) &v2_d, size_v2));
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
/*	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}*/
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d ;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
//	p5ld_v2 = p6d * h3d * p7d;
	h3ld_t3 = 1;
	h1ld_t3 = h3d;
	h2ld_t3 = h1d * h3d;
	p6ld_t3 = h1d * h2d * h3d;
//	p5ld_t3 = p6d * h1d * h2d * h3d;
	p4ld_t3 = p5d * p6d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
//  for(i=0;i<nstreams;++i){

    sd_t_s1_2_kernel<<<dimGrid,dimBlock,0>>>(h1d,h2d,h3d,p4d,p5d*p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t2_d,v2_d,i,total_x, t3_s_d);
		CHECK_ERR("Kernel execution failed");
//	}
/*
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}

	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}*/
	hipDeviceSynchronize();
//	CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));
/*
	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}*/
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
extern          "C" void 
sd_t_s1_2_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d, double *t3, double *t2, double *v2)
{
	sd_t_s1_2_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d,  t3, t2, v2);
}
extern          "C" void 
sd_t_s1_3_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d,  double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double         *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
  /*  if(first==1)
    {
        t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
    }
*/
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d ;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
//	p5ld_v2 = p6d * h3d * p7d;
	h1ld_t3 = 1;
	h3ld_t3 = h1d;
	h2ld_t3 = h1d * h3d;
	p6ld_t3 = h1d * h2d * h3d;
//	p5ld_t3 = p6d * h1d * h2d * h3d;
	p4ld_t3 = p5d * p6d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
  for(i=0;i<nstreams;++i){
    sd_t_s1_1_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d*p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t2_d,v2_d,i,total_x, t3_s_d);
		CHECK_ERR("Kernel execution failed");
	}
/*
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}

	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}
*/	hipDeviceSynchronize();
	//CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));

	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
//	freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
#undef T1
#undef T2
#undef Tcomm
extern          "C" void 
sd_t_s1_3_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d,  double *t3, double *t2, double *v2)
{
	sd_t_s1_3_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d, t3, t2, v2);
}
/*----------------------------------------------------------------------*
 *t3[h3,h2,h1,p6,p4,p5] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
#define T1 16
#define T2 16
#define Tcomm 16
__global__ void sd_t_s1_4_kernel(int h1d,int h2d,int h3d,int p4d,int p5d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int p5ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p5ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
  int h1,h2,h3,p6,p5;
  __shared__ double t2_shm[T1*2*Tcomm];
  
  for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
  if(i<h1d*p4d)
  t2_shm[i] = t2_d[i];
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  rest_x = thread_x;
    __syncthreads();
/* the following computation may need to happen inside the loop */
  for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
  {
    rest_x += i;
  	h3=rest_x%h3d;
  	rest_x=rest_x/h3d;
  	h2=rest_x%h2d;
  	rest_x=rest_x/h2d;
  	p6=rest_x%p6d;
  	rest_x=rest_x/p6d;
  	p5=rest_x%p5d;

    if((thread_x+i)<total_x)
  	for(h1=0;h1<h1d;h1++)
  	for(p4=0;p4<p4d;p4++)
  	{
     	t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p5*p5ld_t3+p4*p4ld_t3]-=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2+p5*p5ld_v2];
  	}
  }
    __syncthreads();
}

extern          "C" void 
sd_t_s1_4_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d,  double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double         *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
  /*  if(first==1)
    {
        t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
    }
*/
//	t3_d = (double *) getGpuMem(size_t3);
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
/*	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}*/
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
	p5ld_v2 = p6d * h3d * h2d;
	h3ld_t3 = 1;
	h2ld_t3 = h3d;
	h1ld_t3 = h2d * h3d;
	p6ld_t3 = h1d * h2d * h3d;
	p4ld_t3 = p6d * h1d * h2d * h3d;
	p5ld_t3 = p4d * p6d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
   i=0;
 // for(i=0;i<nstreams;++i){
    sd_t_s1_4_kernel<<<dimGrid,dimBlock,0>>>(h1d,h2d,h3d,p4d,p5d,p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,p5ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p5ld_t3,p4ld_t3,t3_s_d,t2_d,v2_d,i,total_x);
    //sd_t_s1_4_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d,p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,p5ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p5ld_t3,p4ld_t3,t3_d,t2_d,v2_d,i,total_x);
		CHECK_ERR("Kernel execution failed");
//	}


	hipDeviceSynchronize();
	/*	CUDA_SAFE(hipMemcpy(((char *) t3_p) , ((char *) t3_d) , size_block_t3, hipMemcpyDeviceToHost));
	printf("Time for Async DeviceToHost %f\n", et-st);
	stream = 0;
//	while (stream < nstreams) {
//		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = t3_p; //[stream * size_el_block_t3];
		double         *dst = t3;  //[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] -= src[i];
		}
//		stream++;
//	}
*/
//	hipDeviceSynchronize();
/*
	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}*/
//	freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
#undef T1
#undef T2
#undef Tcomm
extern          "C" void 
sd_t_s1_4_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d, double *t3, double *t2, double *v2)
{
	sd_t_s1_4_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d,  t3, t2, v2);
}

/*----------------------------------------------------------------------*
 *t3[h3,h1,h2,p6,p4,p5] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
#define T1 16
#define T2 16
#define Tcomm 16
__global__ void sd_t_s1_5_kernel(int h1d,int h2d,int h3d,int p4d,int p5d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int p5ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p5ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
  int h1,h2,h3,p6,p5;
  __shared__ double t2_shm[T1*2*Tcomm];
  
  for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
  if(i<h1d*p4d)
  t2_shm[i] = t2_d[i];
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  rest_x = thread_x;
    __syncthreads();
/* the following computation may need to happen inside the loop */
  for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
  {
    rest_x += i;
  	h3=rest_x%h3d;
  	rest_x=rest_x/h3d;
  	h2=rest_x%h2d;
  	rest_x=rest_x/h2d;
  	p6=rest_x%p6d;
  	rest_x=rest_x/p6d;
  	p5=rest_x%p5d;

    if((thread_x+i)<total_x)
  	for(h1=0;h1<h1d;h1++)
  	for(p4=0;p4<p4d;p4++)
  	{
     	t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p5*p5ld_t3+p4*p4ld_t3]+=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2+p5*p5ld_v2];
  	}
  }
    __syncthreads();
}

extern          "C" void 
sd_t_s1_5_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d,  double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double         *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
  /*  if(first==1)
    {
        t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
    }
*/
//	t3_d = (double *) getGpuMem(size_t3);
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d ;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
	p5ld_v2 = p6d * h3d * h2d;
	h3ld_t3 = 1;
	h1ld_t3 = h3d;
	h2ld_t3 = h1d * h3d;
	p6ld_t3 = h1d * h2d * h3d;
	p4ld_t3 = p6d * h1d * h2d * h3d;
	p5ld_t3 = p4d * p6d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
  for(i=0;i<nstreams;++i){
    sd_t_s1_5_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d,p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,p5ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p5ld_t3,p4ld_t3,t3_s_d,t2_d,v2_d,i,total_x);
		CHECK_ERR("Kernel execution failed");
	}
/*
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}

	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}
*/
	hipDeviceSynchronize();

	//CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));
	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
//	freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
#undef T1
#undef T2
#undef Tcomm
extern          "C" void 
sd_t_s1_5_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d,  double *t3, double *t2, double *v2)
{
	sd_t_s1_5_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d,  t3, t2, v2);
}

/*----------------------------------------------------------------------*
 *t3[h1,h3,h2,p6,p4,p5] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
#define T1 16
#define T2 16
#define Tcomm 16
__global__ void sd_t_s1_6_kernel(int h1d,int h2d,int h3d,int p4d,int p5d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int p5ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p5ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
  int h1,h2,h3,p6,p5;
  __shared__ double t2_shm[T1*2*Tcomm];
  
  for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
  if(i<h1d*p4d)
  t2_shm[i] = t2_d[i];
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  rest_x = thread_x;
    __syncthreads();
/* the following computation may need to happen inside the loop */
  for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
  {
    rest_x += i;
  	h3=rest_x%h3d;
  	rest_x=rest_x/h3d;
  	h2=rest_x%h2d;
  	rest_x=rest_x/h2d;
  	p6=rest_x%p6d;
  	rest_x=rest_x/p6d;
  	p5=rest_x%p5d;

    if((thread_x+i)<total_x)
  	for(h1=0;h1<h1d;h1++)
  	for(p4=0;p4<p4d;p4++)
  	{
     	t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p5*p5ld_t3+p4*p4ld_t3]-=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2+p5*p5ld_v2];
  	}
  }
    __syncthreads();
}

extern          "C" void 
sd_t_s1_6_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d,  double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double          *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
  /*  if(first==1)
    {
        t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
    }
*/
//	t3_d = (double *) getGpuMem(size_t3);
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
	p5ld_v2 = p6d * h3d * h2d;
	h1ld_t3 = 1;
	h3ld_t3 = h1d;
	h2ld_t3 = h1d * h3d;
	p6ld_t3 = h1d * h2d * h3d;
	p4ld_t3 = p6d * h1d * h2d * h3d;
	p5ld_t3 = p4d * p6d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
  for(i=0;i<nstreams;++i){
    sd_t_s1_6_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d,p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,p5ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p5ld_t3,p4ld_t3,t3_s_d,t2_d,v2_d,i,total_x);
		CHECK_ERR("Kernel execution failed");
	}
/*	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}

	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}*/
	hipDeviceSynchronize();
	//CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));

	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
//	freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
#undef T1
#undef T2
#undef Tcomm
extern          "C" void 
sd_t_s1_6_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d, double *t3, double *t2, double *v2)
{
	sd_t_s1_6_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d, t3, t2, v2);
}









/*----------------------------------------------------------------------*
 *t3[h3,h2,h1,p4,p6,p5] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
#define T1 16
#define T2 16
#define Tcomm 16
__global__ void sd_t_s1_7_kernel(int h1d,int h2d,int h3d,int p4d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
  int h1,h2,h3,p6;
  __shared__ double t2_shm[T1*2*Tcomm];
  
  for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
  if(i<h1d*p4d)
  t2_shm[i] = t2_d[i];
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  rest_x = thread_x;
    __syncthreads();
/* the following computation may need to happen inside the loop */
  for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
  {
    rest_x += i;
  	h3=rest_x%h3d;
  	rest_x=rest_x/h3d;
  	h2=rest_x%h2d;
  	rest_x=rest_x/h2d;
  	p6=rest_x%p6d;

    if((thread_x+i)<total_x)
  	for(h1=0;h1<h1d;h1++)
  	for(p4=0;p4<p4d;p4++)
  	{
     	t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p4*p4ld_t3]+=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2];
  	}
  }
    __syncthreads();
}
extern          "C" void 
sd_t_s1_7_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d,  double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double         *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
  /*  if(first==1)
    {
        t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
    }
*/
//	t3_d = (double *) getGpuMem(size_t3);
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
//	p5ld_v2 = p6d * h3d * p7d;
	h3ld_t3 = 1;
	h2ld_t3 = h3d;
	h1ld_t3 = h2d * h3d;
	p4ld_t3 = h1d * h2d * h3d;
//	p5ld_t3 = p6d * h1d * h2d * h3d;
	p6ld_t3 = p4d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
  for(i=0;i<nstreams;++i){
    sd_t_s1_7_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d*p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t3_s_d,t2_d,v2_d,i,total_x);
		CHECK_ERR("Kernel execution failed");
	}
/*
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}

	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}*/
	hipDeviceSynchronize();
	//CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));

	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
//	freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
#undef T1
#undef T2
#undef Tcomm
extern          "C" void 
sd_t_s1_7_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d, double *t3, double *t2, double *v2)
{
	sd_t_s1_7_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d, t3, t2, v2);
}
#define T1 16
#define T2 16
#define Tcomm 16
__global__ void sd_t_s1_8_kernel(int h1d,int h2d,int h3d,int p4d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
  int h1,h2,h3,p6;
  __shared__ double t2_shm[T1*2*Tcomm];
  
  for(int i=threadIdx.x;i<h1d*p4d;i+=blockDim.x)
  if(i<h1d*p4d)
  t2_shm[i] = t2_d[i];
  int rest_x=blockIdx.x;
  int thread_x = T2*T1 * rest_x + threadIdx.x;
  rest_x = thread_x;
    __syncthreads();
/* the following computation may need to happen inside the loop */
  for(int i=0;i<total_x;i+=gridDim.x*blockDim.x)
  {
    rest_x += i;
  	h3=rest_x%h3d;
  	rest_x=rest_x/h3d;
  	h2=rest_x%h2d;
  	rest_x=rest_x/h2d;
  	p6=rest_x%p6d;

    if((thread_x+i)<total_x)
  	for(h1=0;h1<h1d;h1++)
  	for(p4=0;p4<p4d;p4++)
  	{
     	t3d[h3*h3ld_t3+h2*h2ld_t3+h1*h1ld_t3+p6*p6ld_t3+p4*p4ld_t3]-=t2_shm[h1*p4d+p4]*v2_d[h3*h3ld_v2+h2*h2ld_v2+p6*p6ld_v2];
  	}
  }
    __syncthreads();
}
/*----------------------------------------------------------------------*
 *t3[h3,h1,h2,p4,p6,p5] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
#define T1 16
#define T2 16
#define Tcomm 16
extern          "C" void 
sd_t_s1_8_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d,  double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double          *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
  /*  if(first==1)
    {
        t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
    }
*/
//	t3_d = (double *) getGpuMem(size_t3);
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
//	p5ld_v2 = p6d * h3d * p7d;
	h3ld_t3 = 1;
	h1ld_t3 = h3d;
	h2ld_t3 = h1d * h3d;
	p4ld_t3 = h1d * h2d * h3d;
//	p5ld_t3 = p6d * h1d * h2d * h3d;
	p6ld_t3 = p4d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
  for(i=0;i<nstreams;++i){
    sd_t_s1_8_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d*p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t3_s_d,t2_d,v2_d,i,total_x);
		CHECK_ERR("Kernel execution failed");
	}
/*
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}
	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}*/
	hipDeviceSynchronize();
//	CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));

	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
//	freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
extern          "C" void 
sd_t_s1_8_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d, double *t3, double *t2, double *v2)
{
	sd_t_s1_8_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d, t3, t2, v2);
}
/*----------------------------------------------------------------------*
 *t3[h1,h3,h2,p4,p6,p5] -= t2[p4,h1] * v2[h3,h2,p6,p5]
 *----------------------------------------------------------------------*/
extern          "C" void 
sd_t_s1_9_cuda(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d,  double *t3, double *t2, double *v2)
{
    double st, et;
//ckbn    st = timer(); 
	size_t          p7ld_t2, p4ld_t2, h1ld_t2, h2ld_v2, p7ld_v2, h3ld_v2,
	                p6ld_v2, p5ld_v2, h3ld_t3, h2ld_t3, h1ld_t3, p6ld_t3,
	                p5ld_t3, p4ld_t3;
	size_t          size_t3, size_block_t3, size_el_block_t3, size_t2,
	                size_v2;
	hipStream_t   *streams;
	size_t          nstreams, i;
	double          *t2_d, *v2_d, *t3_p;
	size_t3 = h3d * h2d * h1d * p6d * p5d * p4d * sizeof(double);
	size_t2 = p4d * h1d * sizeof(double);
	size_v2 = h3d * h2d * p6d * p5d * sizeof(double);
	nstreams = 1;
	size_block_t3 = size_t3 / nstreams;
	size_el_block_t3 = size_block_t3 / sizeof(double);
  /*  if(first==1)
    {
        t3_d = (double *) getGpuMem(size_t3);
        hipMemset(t3_d,0,size_t3*sizeof(double));
        first = 0;
    }
*/
//	t3_d = (double *) getGpuMem(size_t3);
	t2_d = (double *) getGpuMem(size_t2);
	v2_d = (double *) getGpuMem(size_v2);
	t3_p = (double *) getHostMem(size_t3);
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	assert(streams != NULL);
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipStreamCreate(&streams[i]));
	}
	CUDA_SAFE(hipMemcpy(t2_d, t2, size_t2, hipMemcpyHostToDevice));
	CUDA_SAFE(hipMemcpy(v2_d, v2, size_v2, hipMemcpyHostToDevice));

	p4ld_t2 = 1;
	h1ld_t2 = p4d;

	h3ld_v2 = 1;
	h2ld_v2 = h3d;
	p6ld_v2 = h3d * h2d;
//	p5ld_v2 = p6d * h3d * p7d;
	h1ld_t3 = 1;
	h3ld_t3 = h1d;
	h2ld_t3 = h1d * h3d;
	p4ld_t3 = h1d * h2d * h3d;
//	p5ld_t3 = p6d * h1d * h2d * h3d;
	p6ld_t3 = p4d * h1d * h2d * h3d;
  int total_x = h3d*h2d*p6d*p5d;
  dim3 dimBlock(T2*T1);dim3 dimGrid(DIV_UB(total_x,T2*T1), 1);
  for(i=0;i<nstreams;++i){
    sd_t_s1_7_kernel<<<dimGrid,dimBlock,0,streams[i]>>>(h1d,h2d,h3d,p4d,p5d*p6d,p4ld_t2,h1ld_t2,h3ld_v2,h2ld_v2,p6ld_v2,h3ld_t3,h2ld_t3,h1ld_t3,p6ld_t3,p4ld_t3,t3_s_d,t2_d,v2_d,i,total_x);
		CHECK_ERR("Kernel execution failed");
	}
/*
	for (i = 0; i < nstreams; ++i) {
		CUDA_SAFE(hipMemcpyAsync(((char *) t3_p) + i * size_block_t3, ((char *) t3_s_d) + i * size_block_t3, size_block_t3, hipMemcpyDeviceToHost, streams[i]));
	}
	stream = 0;
	while (stream < nstreams) {
		while (hipStreamQuery(streams[stream]) != hipSuccess);
		double         *src = &t3_p[stream * size_el_block_t3];
		double         *dst = &t3[stream * size_el_block_t3];
		for (i = 0; i < size_el_block_t3; ++i) {
			dst[i] = src[i];
		}
		stream++;
	}*/
	hipDeviceSynchronize();
	//CUDA_SAFE(hipMemcpy(((char *) t3) , ((char *) t3_s_d) , size_t3, hipMemcpyDeviceToHost));

//  printf("out is %lf\n", t3_p[0]);
	for (i = 0; i < nstreams; ++i) {
		hipStreamDestroy(streams[i]);
	}
	//freeGpuMem(t3_d);
	freeGpuMem(t2_d);
	freeGpuMem(v2_d);
	freeHostMem(t3_p);
	free(streams);
}
extern          "C" void 
sd_t_s1_9_cuda_(Integer * h1d, Integer * h2d, Integer * h3d, Integer * p4d, Integer * p5d, Integer * p6d,  double *t3, double *t2, double *v2)
{
	sd_t_s1_9_cuda((int) *h1d, (int) *h2d, (int) *h3d, (int) *p4d, (int) *p5d, (int) *p6d,  t3, t2, v2);
}
